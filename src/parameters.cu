#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>

using namespace std;

int contadorll = 0;
//extern bool mousefree;
extern GLint windowWidth;
extern GLint windowHeight;
///extern string fps;
//map<int, map<string, map<string, GLfloat> > > atom_statistic;
map<string, string> tw;

//class GUI: public QMainWindow {
//public:
//	GUI();
//}
void displayMe(void) {
	contadorll++;
	printf("A %d\n", contadorll);
	glClear(GL_COLOR_BUFFER_BIT);
	glBegin(GL_POLYGON);
	glVertex3f(0.0, 0.0, 0.0);
	glVertex3f(0.5, 0.0, 0.0);
	glVertex3f(0.5, 0.5, 0.0);

//	glMatrixMode(GL_PROJECTION);
//	glPushMatrix();
//	glLoadIdentity();
//	gluOrtho2D(0.0, 300, 0.0, 300);
//	glMatrixMode(GL_MODELVIEW);
//	glPushMatrix();
//	glLoadIdentity();
//	glColor3f(0.0, 1.0, 0.0); // Green
//	glRasterPos2i(10, 10);
//	void * font = GLUT_BITMAP_9_BY_15;
//	stringstream ss;
//	ss << contadorll;
//	string texto = ss.str();
//	for (string::iterator i = texto.begin(); i != texto.end(); ++i) {
//		char c = *i;
//		glutBitmapCharacter(font, c);
//	}
//	glMatrixMode(GL_MODELVIEW);
//	glPopMatrix();
//
//	glMatrixMode(GL_PROJECTION);
//	glPopMatrix();
//	// ----- Stop Drawing Stuff! ------
//	glfwSwapBuffers();

	glEnd();
	glFlush();
}

void help(){
	tw["Campo1"] = "Teste1";
	tw["Campo2"] = "Teste2";
	tw["Campo3"] = "Teste3";
	tw["Campo4"] = "Teste4";
}


void parameters_window() {
	help();
	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();
	gluOrtho2D(0.0, windowWidth, 0.0, windowHeight);
	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();
	glLoadIdentity();
	glColor3f(0.0, 1.0, 0.0);				// Green
	GLint distancia_topo = windowHeight;
	string texto = "";
	void * font = GLUT_BITMAP_9_BY_15;
	typedef map<string, string>::iterator it_type;
	for(it_type iterator = tw.begin(); iterator != tw.end(); iterator++) {
		distancia_topo -= 14;
	    glRasterPos2i(10, distancia_topo);
		texto = iterator->first+" = "+iterator->second;
		for (string::iterator i = texto.begin(); i != texto.end(); ++i) {
			char c = *i;
			glutBitmapCharacter(font, c);
		}
	}
	glMatrixMode(GL_MODELVIEW);
	glPopMatrix();

	glMatrixMode(GL_PROJECTION);
	glPopMatrix();
}

void handleKeypress(int theKey, int theAction) {
/*
	if (theAction == GLFW_PRESS) {
		if (pressionando_control) {
			switch (theKey) {
			case 'C':
				add_atom('C');
				break;

			case 'H':
				add_atom('H');
				break;

			case 'O':
				add_atom('O');
				break;

			case 'N':
				add_atom('N');
				break;

			case '1':
				ativa_desativa_colisao_tensao();
				break;

			case '2':
				ativa_desativa_tensao_hb();
				break;

			case '3':
				compare_protein_build_MD(0);
				break;

			case '4':
				ativa_desativa_comparation();
				break;

			case '5':
				restaura_posicoes();
				break;

			case '6':
				ativa_desativa_comparation_speed();
				break;

			case '.':
				calibration_precision++;
				distance_calibration();
				break;

			case ',':
				calibration_precision--;
				distance_calibration();
				break;
			}
		} else {
			switch (theKey) {
			case 289:
				pressionando_control = true;
				break;

			case 'W':
				holdingForward = true;
				break;

			case 'S':
				holdingBackward = true;
				break;

			case 'A':
				holdingLeftStrafe = true;
				break;

			case 'D':
				holdingRightStrafe = true;
				break;

			case 'K':
				pressionando_k = true;
				break;

			case 'J':
				pressionando_j = true;
				break;

			case 'I':
				pressionando_i = true;
				break;

			case 'M':
				pressionando_m = true;
				break;

			case 'Y':
				posz[0] -= 0.5;
				break;

			case 'H':
				posz[0] += 0.5;
				break;

			case 'O':
				pressionando_o = true;
				break;

			case 'P':
				pressionando_p = true;
				break;

			case '0':
				pressionando_0 = true;
				break;

			case 'L':
				pressionando_l = true;
				break;

			case '8':
				pressionando_8 = true;
				break;

			case '9':
				pressionando_9 = true;
				break;

			case 'B':
				show_variables();
				break;

			case 'G':
				add_chain();
				break;

			case '1':
				camera_position(0.0, 0.0, caixa_tamanho, 0.0, 0.0, 0.0);
				break;

			case '2':
				camera_position(caixa_tamanho * 6.0, 0.0, -90.0, 0.0, -90.0, 0.0);
				break;

			case '3':
				camera_position(0.0, caixa_tamanho * 6.0, -90.0, 90.0, 180.0, -90.0);
				break;

			case '4':
				camera_position(caixa_tamanho * 3.0, caixa_tamanho * 3.0, caixa_tamanho, 27.0, -30.0, 0.0);
				break;

			case 'E':
				ativa_desativa_perimetro();
				break;

			case 'R':
				ativa_desativa_rastreio();
				break;

			case 'F':
				ativa_desativa_forca();
				break;

			case 'Z':
				rem_energy();
				break;

			case 'X':
				add_energy();
				break;

			case 'C':
				continua();
				break;

			case ',':
				caixa_tamanho -= 0.5;
				break;

			case '.':
				caixa_tamanho += 0.5;
				break;

			case 'T':
				ativa_desativa_base();
				break;

			case '5':
				ativa_desativa_base_line();
				break;

			default:
				// Do nothing...
				break;
			}
		}
	} else // If a key is released, toggle the relevant key-release flag
	{
		switch (theKey) {
		case 289:
			pressionando_control = false;
			break;

		case 'W':
			holdingForward = false;
			break;

		case 'S':
			holdingBackward = false;
			break;

		case 'A':
			holdingLeftStrafe = false;
			break;

		case 'D':
			holdingRightStrafe = false;
			break;

		case 'K':
			pressionando_k = false;
			break;

		case 'J':
			pressionando_j = false;
			break;

		case 'I':
			pressionando_i = false;
			break;

		case 'M':
			pressionando_m = false;
			break;

		case 'O':
			pressionando_o = false;
			break;

		case 'P':
			pressionando_p = false;
			break;

		case '0':
			pressionando_0 = false;
			break;

		case 'L':
			pressionando_l = false;
			break;

		case '8':
			pressionando_8 = false;
			break;

		case '9':
			pressionando_9 = false;
			break;

		default:
			// Do nothing...
			break;
		}
	}
	*/
}
