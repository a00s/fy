#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include "gvariaveis.h"

using namespace std;
GLint menu_page = 0;

extern map<string, map<string, GLfloat*> > vF;
extern map<string, map<string, GLint*> > vI;
extern map<string, map<string, bool*> > vB;

//extern map<string, GLfloat> vs;
//extern map<string, map<string, map<string, map<string, map<string, map<string, GLfloat> > > > > > vp;

int contadorll = 0;
map<string, string> tw;

// Variavel de sistema
//void setVS(string s1, GLfloat valor) {
//	vs[s1] = valor;
//}

//GLfloat getVS(string s1) {
//	return 0.0;
//	return vs[s1];
//}

void parameters_window() {

	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();
	gluOrtho2D(0.0, windowWidth, 0.0, windowHeight);
	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();
	glLoadIdentity();
	if (menu_page == 0) {
		glColor3f(0.0, 1.0, 0.0);
	} else if (menu_page == 1) {
		glColor3f(1.0, 1.0, 0.0);
	} else if (menu_page == 2) {
		glColor3f(0.0, 1.0, 1.0);
	} else if (menu_page == 3) {
		glColor3f(1.0, 1.0, 1.0);
	} else if (menu_page == 4) {
		glColor3f(0.0f, 1.0f, 0.0f);
	} else {
		glColor3f(0.0, 1.0, 0.0);				// Green
	}
	GLint distancia_topo = windowHeight;
	string texto = "";
	void * font = GLUT_BITMAP_9_BY_15;

	typedef map<string, string>::iterator it_type;
	for (it_type iterator = tw.begin(); iterator != tw.end(); iterator++) {
		distancia_topo -= 14;
		glRasterPos2i(10, distancia_topo);
		texto = iterator->first + " = " + iterator->second;
		for (string::iterator i = texto.begin(); i != texto.end(); ++i) {
			char c = *i;
			glutBitmapCharacter(font, c);
		}
	}
	glMatrixMode(GL_MODELVIEW);
	glPopMatrix();

	glMatrixMode(GL_PROJECTION);
	glPopMatrix();
}

//string Convert(GLfloat number) {
//	ostringstream buff;
//	buff << number;
//	return buff.str();
//}

string Convert(GLfloat *number) {
	ostringstream buff;
	buff << *number;
	return buff.str();
}

string Convert(GLint *number) {
	ostringstream buff;
	buff << *number;
	return buff.str();
}

string Convert(bool *number) {
	ostringstream buff;
	buff << *number;
	return buff.str();
}

void imprime_menu(string tipomenu) {
	tw.clear();
//	tw[""] = "Variaveis de Screen";
	for (map<string, map<string, GLfloat*> >::iterator i = vF.begin(); i != vF.end(); ++i) {
		if ((*i).first == tipomenu) {
			for (map<string, GLfloat*>::iterator ii = i->second.begin(); ii != i->second.end(); ++ii) {
				string campochave = (*ii).first;
				tw[campochave] = Convert((*ii).second);
			}
		}
	}
	for (map<string, map<string, GLint*> >::iterator i = vI.begin(); i != vI.end(); ++i) {
		if ((*i).first == tipomenu) {
			for (map<string, GLint*>::iterator ii = i->second.begin(); ii != i->second.end(); ++ii) {
				string campochave = (*ii).first;
				tw[campochave] = Convert((*ii).second);
			}
		}
	}
	for (map<string, map<string, bool*> >::iterator i = vB.begin(); i != vB.end(); ++i) {
		if ((*i).first == tipomenu) {
			for (map<string, bool*>::iterator ii = i->second.begin(); ii != i->second.end(); ++ii) {
				string campochave = (*ii).first;
				tw[campochave] = Convert((*ii).second);
				if (Convert((*ii).second) == "1") {
					tw[campochave] = "true";
				} else {
					tw[campochave] = "false";
				}
			}
		}
	}
}

void handleKeypress(int theKey, int theAction) {
	if (theAction == GLFW_PRESS) {
		// ------------- Padrao ------------------
		switch (theKey) {
		case 289:
			pressionando_control = true;
			break;
		case '0':
			menu_page = 0;
			break;
		case '1':
			menu_page = 1;
			break;
		case '2':
			menu_page = 2;
			break;
		case '3':
			menu_page = 3;
			break;
		case '4':
			menu_page = 4;
			break;
		case 'W':
			holdingForward = true;
			break;
		case 'S':
			holdingBackward = true;
			break;
		case 'A':
			holdingLeftStrafe = true;
			break;
		case 'D':
			holdingRightStrafe = true;
			break;
		default:
			break;
		}
	} else // If a key is released, toggle the relevant key-release flag
	{
		switch (theKey) {
		case 289:
			pressionando_control = false;
			break;

		case 'W':
			holdingForward = false;
			break;

		case 'S':
			holdingBackward = false;
			break;

		case 'A':
			holdingLeftStrafe = false;
			break;

		case 'D':
			holdingRightStrafe = false;
			break;
		}
	}

	if (menu_page == 0) {
		tw.clear();
		tw["0"] = "Menu";
		tw["1"] = "Navegacao";
		tw["2"] = "Alteracoes";
		tw["3"] = "Variaveis de tela";
		tw["4"] = "Variaveis";
		tw["-"] = "-------------";
		tw["W"] = "Sobe";
		tw["S"] = "Desce";
		tw["A"] = "Esquerda";
		tw["D"] = "Direita";
	} else if (menu_page == 1) {
		// Navegacao
		tw.clear();
		tw["P"] = "Restaura posicoes";
		tw["C"] = "Compara Protein build MD";
		tw["V"] = "A/D comparacao speed";
		tw["X"] = "A/D comparacao";
		tw["E"] = "Camera 1";
		tw["R"] = "Camera 2";
		tw["T"] = "Camera 3";
		tw["Y"] = "Camera 4";
		tw[","] = "Diminui caixa";
		tw["."] = "Aumenta caixa";
		tw["B"] = "A/D base";
		tw["N"] = "A/D linha base";
		tw["M"] = "A/D perimetro";
		tw["J"] = "A/D rastreio";
		tw["F"] = "A/D forca";
		tw["G"] = "A/D ghost protein";
		if (theAction == GLFW_PRESS) {
			switch (theKey) {
			case 'P':
				restaura_posicoes();
				break;
			case 'C':
				compare_protein_build_MD(0);
				break;
			case 'V':
				ativa_desativa_comparation_speed();
				break;
			case 'X':
				ativa_desativa_comparation();
				break;
			case 'E':
				camera_position(0.0, 0.0, -15.0, 0.0, 0.0, 0.0);
				break;
			case 'R':
				camera_position(-40.0, 0.0, -50.0, 0.0, 90.0, 0.0);
				break;
			case 'T':
				camera_position(0.0, 40.0, -50.0, 90.0, 0.0, 0.0);
				break;
			case 'Y':
				camera_position(caixa_tamanho * 3.0, caixa_tamanho * 3.0, caixa_tamanho, 27.0, -30.0, 0.0);
				break;
			case ',':
				caixa_tamanho -= 0.5;
				break;
			case '.':
				caixa_tamanho += 0.5;
				break;
			case 'B':
				ativa_desativa_base();
				break;
			case 'N':
				ativa_desativa_base_line();
				break;
			case 'M':
				ativa_desativa_perimetro();
				break;
			case 'J':
				ativa_desativa_rastreio();
				break;
			case 'F':
				ativa_desativa_forca();
				break;
			case 'G':
				ativa_desativa_ghost_protein();
				break;
			case 'I':
				angulo_adicional_teste -= 1.0;
//				posx[36] -= 0.1;
				break;
			case 'O':
				angulo_adicional_teste += 1.0;
//				posx[36] += 0.1;
				break;
			case '9':
				posy[36] += 0.1;
				break;
			case 'K':
				posy[36] -= 0.1;
				break;
			case 'L':
				posz[36] -= 0.1;
				break;
			case ';':
				posz[36] += 0.1;
				break;
			case '[':
				sequencial_mostra--;
				break;
			case ']':
				sequencial_mostra++;
				break;
			}
		} else {

		}
	} else if (menu_page == 2) {
		// Alteracoes
		tw.clear();
		tw["T"] = "A/D tensao de colisao";
		tw["H"] = "A/D tensao de HB";
		tw["."] = "+ tune distancia";
		tw[","] = "- tune distancia";
		tw["Z"] = "+ energia";
		tw["X"] = "- energia";
		tw["P"] = "Pausa/Continua";
		tw["G"] = "Adiciona aminoacido";
		tw["K"] = "+X";
		tw["J"] = "-X";
		tw["I"] = "+Y";
		tw["M"] = "-Y";

		if (theAction == GLFW_PRESS) {
			switch (theKey) {
			case 'T':
				ativa_desativa_colisao_tensao();
				break;
			case 'H':
				ativa_desativa_tensao_hb();
				break;
			case '.':
				calibration_precision++;
				distance_calibration();
				break;
			case ',':
				calibration_precision--;
				distance_calibration();
				break;
			case 'Z':
				rem_energy();
				break;
			case 'X':
				add_energy();
				break;
			case 'P':
				continua();
				break;
			case 'G':
				add_chain();
				break;
				// X+
			case 'K':
				pressionando_k = false;
				break;
				// X-
			case 'J':
				pressionando_j = false;
				break;
				// Y+
			case 'I':
				pressionando_i = false;
				break;
				// Y-
			case 'M':
				pressionando_m = false;
				break;
			}
		} else {

		}
	} else if (menu_page == 3) {
		imprime_menu("screen");
	} else if (menu_page == 4) {
		// Variaveis
		imprime_menu("tuning");
//		tw.clear();
//		tw[""] = "Propriedades";

//		for (map<string, map<string, map<string, map<string, map<string, map<string, GLfloat> > > > > >::iterator i = vp.begin(); i != vp.end(); ++i) {
//			if ((*i).first != "PDBcalibrationMin" && (*i).first != "PDBcalibrationMax") {
//				for (map<string, map<string, map<string, map<string, map<string, GLfloat> > > > >::iterator ii = i->second.begin(); ii != i->second.end(); ++ii) {
//					for (map<string, map<string, map<string, map<string, GLfloat> > > >::iterator iii = ii->second.begin(); iii != ii->second.end(); ++iii) {
//						for (map<string, map<string, map<string, GLfloat> > >::iterator iiii = iii->second.begin(); iiii != iii->second.end(); ++iiii) {
//							for (map<string, map<string, GLfloat> >::iterator iiiii = iiii->second.begin(); iiiii != iiii->second.end(); ++iiiii) {
//								for (map<string, GLfloat>::iterator iiiiii = iiiii->second.begin(); iiiiii != iiiii->second.end(); ++iiiiii) {
//									string campochave = (*i).first + ' ' + (*ii).first + ' ' + (*iii).first + ' ' + (*iiii).first + ' ' + (*iiiii).first + ' ' + (*iiiiii).first;
//									tw[campochave] = Convert((*iiiiii).second);
//								}
//							}
//						}
//					}
//				}
//			}
//		}
	}
//	if (pressionando_control) {
//		switch (theKey) {
//		case 'C':
//			add_atom('C');
//			break;
//
//		case 'H':
//			add_atom('H');
//			break;
//
//		case 'O':
//			add_atom('O');
//			break;
//
//		case 'N':
//			add_atom('N');
//			break;
//

}
