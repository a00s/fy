#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include "mysql_connection.h"
#include <cppconn/driver.h>
#include <cppconn/exception.h>
#include <cppconn/resultset.h>
#include <cppconn/statement.h>
#include <cppconn/prepared_statement.h>
using namespace std;

extern GLint calibration_precision;
extern GLint calibration_precision_out;
extern GLfloat calibrationMin[20][34][2][20][34]; // [Amino1][Atom1][Same|Another][Amino2][Atom2]
extern GLfloat calibrationMax[20][34][2][20][34]; // [Amino1][Atom1][Same|Another][Amino2][Atom2]

int get_amino_number(const char *amino_sigla) {
	//  Glu = 0
	//  Tyr = 1
	//  Gln = 2
	//  Ile = 3
	//  Trp = 4
	//  Lys = 5
	//  Leu = 6
	//	Ala = 7
	//	Arg = 8
	//	Asp = 9
	//	Ser = 10
	//	Thr = 11
	//	Gly = 12
	//	Met = 13
	//	Phe = 14
	//	Val = 15
	//	Asn = 16
	//	Cys = 17
	//	His = 18
	//	Pro = 19

	if (!strcmp(amino_sigla, "GLU")) {
		return 0;
	} else if (!strcmp(amino_sigla, "TYR")) {
		return 1;
	} else if (!strcmp(amino_sigla, "GLN")) {
		return 2;
	} else if (!strcmp(amino_sigla, "ILE")) {
		return 3;
	} else if (!strcmp(amino_sigla, "TRP")) {
		return 4;
	} else if (!strcmp(amino_sigla, "LYS")) {
		return 5;
	} else if (!strcmp(amino_sigla, "LEU")) {
		return 6;
	} else if (!strcmp(amino_sigla, "ALA")) {
		return 7;
	} else if (!strcmp(amino_sigla, "ARG")) {
		return 8;
	} else if (!strcmp(amino_sigla, "ASP")) {
		return 9;
	} else if (!strcmp(amino_sigla, "SER")) {
		return 10;
	} else if (!strcmp(amino_sigla, "THR")) {
		return 11;
	} else if (!strcmp(amino_sigla, "GLY")) {
		return 12;
	} else if (!strcmp(amino_sigla, "MET")) {
		return 13;
	} else if (!strcmp(amino_sigla, "PHE")) {
		return 14;
	} else if (!strcmp(amino_sigla, "VAL")) {
		return 15;
	} else if (!strcmp(amino_sigla, "ASN")) {
		return 16;
	} else if (!strcmp(amino_sigla, "CYS")) {
		return 17;
	} else if (!strcmp(amino_sigla, "HIS")) {
		return 18;
	} else if (!strcmp(amino_sigla, "PRO")) {
		return 19;
	}
	return 99;
}

int get_atom_number(const char *atomo_letra_local) {
	if (!strcmp(atomo_letra_local, "C1")) {
		return 0;
	} else if (!strcmp(atomo_letra_local, "C2")) {
		return 1;
	} else if (!strcmp(atomo_letra_local, "C3")) {
		return 2;
	} else if (!strcmp(atomo_letra_local, "C4")) {
		return 3;
	} else if (!strcmp(atomo_letra_local, "C5")) {
		return 4;
	} else if (!strcmp(atomo_letra_local, "C6")) {
		return 5;
	} else if (!strcmp(atomo_letra_local, "C7")) {
		return 6;
	} else if (!strcmp(atomo_letra_local, "C8")) {
		return 7;
	} else if (!strcmp(atomo_letra_local, "C9")) {
		return 8;
	} else if (!strcmp(atomo_letra_local, "C10")) {
		return 9;
	} else if (!strcmp(atomo_letra_local, "C11")) {
		return 10;
	} else if (!strcmp(atomo_letra_local, "N1")) {
		return 11;
	} else if (!strcmp(atomo_letra_local, "N2")) {
		return 12;
	} else if (!strcmp(atomo_letra_local, "N3")) {
		return 13;
	} else if (!strcmp(atomo_letra_local, "N4")) {
		return 14;
	} else if (!strcmp(atomo_letra_local, "O1")) {
		return 15;
	} else if (!strcmp(atomo_letra_local, "O2")) {
		return 16;
	} else if (!strcmp(atomo_letra_local, "O3")) {
		return 17;
	} else if (!strcmp(atomo_letra_local, "O4")) {
		return 18;
	} else if (!strcmp(atomo_letra_local, "H1")) {
		return 19;
	} else if (!strcmp(atomo_letra_local, "H2")) {
		return 20;
	} else if (!strcmp(atomo_letra_local, "H3")) {
		return 21;
	} else if (!strcmp(atomo_letra_local, "H4")) {
		return 22;
	} else if (!strcmp(atomo_letra_local, "H5")) {
		return 23;
	} else if (!strcmp(atomo_letra_local, "H6")) {
		return 24;
	} else if (!strcmp(atomo_letra_local, "H7")) {
		return 25;
	} else if (!strcmp(atomo_letra_local, "H8")) {
		return 26;
	} else if (!strcmp(atomo_letra_local, "H9")) {
		return 27;
	} else if (!strcmp(atomo_letra_local, "H10")) {
		return 28;
	} else if (!strcmp(atomo_letra_local, "H11")) {
		return 29;
	} else if (!strcmp(atomo_letra_local, "H12")) {
		return 30;
	} else if (!strcmp(atomo_letra_local, "H13")) {
		return 31;
	} else if (!strcmp(atomo_letra_local, "H14")) {
		return 32;
	} else if (!strcmp(atomo_letra_local, "S1")) {
		return 33;
	}
	return 0;
}

void distance_calibration() {
	printf("Calibrando a partir do mysql\n");
	try {
		sql::Driver *driver;
		sql::Connection *con;
		sql::PreparedStatement *pstmt;
		sql::ResultSet *res;
		driver = get_driver_instance();
		con = driver->connect("tcp://127.0.0.1:3306", "a00s_230", "testando");
		con->setSchema("a00s_230");
		printf("Calibration precision %d\n", calibration_precision);
		printf("Aqui 0.3\n");
		// ------ Dictionary -------
		// Same = 0 / Different = 1
		// Glu = 0
		// Tyr = 1
		// Gln = 2
		// Ile = 3
		// Trp = 4
		// Lys = 5

		// C1 = 0
		// C2 = 1
		// C3 = 2
		// C4 = 3
		// C5 = 4
		// C6 = 5
		// C7 = 6
		// C8 = 7
		// C9 = 8
		// C10 = 9
		// C11 = 10

		// N1 = 11
		// N2 = 12
		// N3 = 13
		// N4 = 14

		// O1 = 15
		// O2 = 16
		// O3 = 17
		// O4 = 18

		// H1 = 19
		// H2 = 20
		// H3 = 21
		// H4 = 22
		// H5 = 23
		// H6 = 24
		// H7 = 25
		// H8 = 26
		// H9 = 27
		// H10 = 28
		// H11 = 29
		// H12 = 30
		// H13 = 31
		// H14 = 32

		// S1 = 33

		// [Amino1][Atom1][Same|Different][Amino2][Atom2]
		// -------- Same amino acid ---------
//		SELECT * FROM (SELECT * FROM a_380484 WHERE i_380512 = 1 ORDER BY i_380529 DESC) tabtemp GROUP BY i_380488,i_380500,i_380494,i_380506
//		pstmt = con->prepareStatement("SELECT i_380488 amino, i_380494 atom1, i_380506 atom2, i_380517 min_distance, i_380523 max_distance FROM a_380484 WHERE i_380529=? AND i_380512=1");
		pstmt = con->prepareStatement("SELECT i_380488 amino, i_380494 atom1, i_380506 atom2, i_380517 min_distance, i_380523 max_distance FROM (SELECT * FROM a_380484 WHERE i_380512 = 1 ORDER BY i_380529 DESC) tabtemp GROUP BY i_380488,i_380500,i_380494,i_380506");
//		pstmt->setInt(1, calibration_precision);
		res = pstmt->executeQuery();
		while (res->next()) {
			calibrationMin[get_amino_number(res->getString("amino").c_str())][get_atom_number(res->getString("atom1").c_str())][0][get_amino_number(res->getString("amino").c_str())][get_atom_number(res->getString("atom2").c_str())] = res->getDouble(4);
			calibrationMax[get_amino_number(res->getString("amino").c_str())][get_atom_number(res->getString("atom1").c_str())][0][get_amino_number(res->getString("amino").c_str())][get_atom_number(res->getString("atom2").c_str())] = res->getDouble(5);
		}
		// -------- Different amino acid ---------
//		pstmt = con->prepareStatement("SELECT i_380488 aminofrom, i_380500 aminoto, i_380494 atom1, i_380506 atom2, i_380517 min_distance, i_380523 max_distance FROM a_380484 WHERE i_380529=? AND i_380512 IS NULL");
		pstmt = con->prepareStatement("SELECT i_380488 aminofrom, i_380500 aminoto, i_380494 atom1, i_380506 atom2, i_380517 min_distance, i_380523 max_distance FROM (SELECT * FROM a_380484 WHERE i_380512 IS NULL ORDER BY i_380529 DESC) tabtemp GROUP BY i_380488,i_380500,i_380494,i_380506");
//		pstmt->setInt(1, calibration_precision_out);
		res = pstmt->executeQuery();
		while (res->next()) {
			calibrationMin[get_amino_number(res->getString("aminofrom").c_str())][get_atom_number(res->getString("atom1").c_str())][1][get_amino_number(res->getString("aminoto").c_str())][get_atom_number(res->getString("atom2").c_str())] = res->getDouble(5);
		}
		delete res;
		delete pstmt;
		delete con;
		printf("System calibrated\n");
	} catch (sql::SQLException &e) {
		printf("%s", e.getErrorCode());
	}
}
