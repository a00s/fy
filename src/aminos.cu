#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include "mysql_connection.h"
#include <cppconn/driver.h>
#include <cppconn/exception.h>
#include <cppconn/resultset.h>
#include <cppconn/prepared_statement.h>
#include <cppconn/statement.h>
using namespace std;

vector<GLfloat> pdbamin_coordenadas_x;
vector<GLfloat> pdbamin_coordenadas_y;
vector<GLfloat> pdbamin_coordenadas_z;

struct VectorProtein {
	GLint aminoseq;
	string label;
	GLfloat x;
	GLfloat y;
	GLfloat z;
};
vector<VectorProtein> pdb_analise;

//struct StructAmino {
//	GLfloat x;
//	GLfloat y;
//	GLfloat z;
//};
vector<VectorProtein> AminoTemp1;
vector<VectorProtein> AminoTemp2;

vector<VectorProtein> AminoTemp1MD;
vector<VectorProtein> AminoTemp2MD;

struct VectorDistMinMax {
	GLint aminoseq;
	string label;
	GLfloat min;
	GLfloat max;
};
vector<VectorDistMinMax> CalcMinMax;
vector<VectorDistMinMax> CalcMinMaxMD;

GLint cont_amin = 0;
extern int atomos_quantidade;
extern char atomo_letra[1000];
extern GLfloat nucleo_proximity[1000];
extern GLfloat nucleo_proximity_free[1000];
extern GLfloat nucleo_proximity_HB[1000];
extern GLint electron_quantidade[1000];
extern GLfloat electron_raio[1000];
extern GLfloat electron_raio_HB[1000];
extern GLint electron_arested[1000][4][2];
extern GLfloat massa[1000];
extern GLfloat posx[1000];
extern GLfloat posy[1000];
extern GLfloat posz[1000];
extern GLfloat velocidade_x[1000]; // velocidade
extern GLfloat velocidade_y[1000]; // velocidade
extern GLfloat velocidade_z[1000]; // velocidade
extern GLfloat electron_y[1000][4];
extern GLfloat electron_z[1000][4];
extern GLint amino[1000];
extern GLint amino_sequencial[1000];
extern GLint atomo_letraN[1000];
extern string atomo_letraL[1000];
extern GLint contador_amino;
extern int get_amino_number(const char *amino_sigla);
extern int get_atom_number(const char *atomo_letra_local);
extern bool atomo_base[1000];
extern GLint chain_ultimo_atomo;
extern map<int, map<string, map<string, GLfloat> > > atom_statistic;
map<string, int> mapa_atomo;
map<string, int> electron_contador;

void connect_electron(string atomo1, string atomo2) {
	electron_arested[mapa_atomo[atomo1]][electron_contador[atomo1]][0] = mapa_atomo[atomo2];
	electron_arested[mapa_atomo[atomo1]][electron_contador[atomo1]][1] = electron_contador[atomo2];
	electron_arested[mapa_atomo[atomo2]][electron_contador[atomo2]][0] = mapa_atomo[atomo1];
	electron_arested[mapa_atomo[atomo2]][electron_contador[atomo2]][1] = electron_contador[atomo1];
	electron_contador[atomo1]++;
	electron_contador[atomo2]++;
}

void add_atoml(string atomo_ll) {
	if (atomos_quantidade == 998) {
		printf("Quantidade de atomos chegou no limite \n");
		return;
	}

	if (atomo_ll == "C") {
		atomo_letra[atomos_quantidade] = 'C';
		nucleo_proximity[atomos_quantidade] = 1.7;
		nucleo_proximity_free[atomos_quantidade] = 3.4; // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 4;
		electron_raio[atomos_quantidade] = 3.4;
		massa[atomos_quantidade] = 12.0107;
	} else if (atomo_ll == "H") {
		atomo_letra[atomos_quantidade] = 'H';
		nucleo_proximity[atomos_quantidade] = 1.2;
		nucleo_proximity_free[atomos_quantidade] = 2.4;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 1;
		electron_raio[atomos_quantidade] = 2.4;
		electron_raio_HB[atomos_quantidade] = 2.4;
		massa[atomos_quantidade] = 1.0079;
		nucleo_proximity_HB[atomos_quantidade] = 1.2;

//		electron_y_HB[atomos_quantidade] = 270;
	} else if (atomo_ll == "O") {
		atomo_letra[atomos_quantidade] = 'O';
		nucleo_proximity[atomos_quantidade] = 1.52;
		nucleo_proximity_free[atomos_quantidade] = 3.04;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 2;
		electron_raio[atomos_quantidade] = 3.04;
		electron_raio_HB[atomos_quantidade] = 3.04;
		massa[atomos_quantidade] = 15.099;
		nucleo_proximity_HB[atomos_quantidade] = 1.52;

//		electron_y_HB[atomos_quantidade] = 90;
	} else if (atomo_ll == "N") {
		atomo_letra[atomos_quantidade] = 'N';
//		nucleo_proximity[atomos_quantidade] = 1.55; // original
		nucleo_proximity[atomos_quantidade] = 1.46; // average collected
		nucleo_proximity_free[atomos_quantidade] = 3.1;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 4;
		electron_raio[atomos_quantidade] = 2.92;
		electron_raio_HB[atomos_quantidade] = 3.1;
		massa[atomos_quantidade] = 14.0067;
		nucleo_proximity_HB[atomos_quantidade] = 1.55;
	} else if (atomo_ll == "S") {
		atomo_letra[atomos_quantidade] = 'S';
		nucleo_proximity[atomos_quantidade] = 1.8;
		nucleo_proximity_free[atomos_quantidade] = 3.6;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 4;
		electron_raio[atomos_quantidade] = 3.6;
		massa[atomos_quantidade] = 32.06;
	}
	posx[atomos_quantidade] = 1.0;
	posy[atomos_quantidade] = 3.0;
	posz[atomos_quantidade] = 2.0;
	velocidade_x[atomos_quantidade] = 0.0;
	velocidade_y[atomos_quantidade] = 0.0;
	velocidade_z[atomos_quantidade] = 0.0;
	electron_y[atomos_quantidade][0] = 345.0;
	electron_z[atomos_quantidade][0] = 350.0;

	electron_y[atomos_quantidade][1] = 100.0;
	electron_z[atomos_quantidade][1] = 150.0;

	electron_y[atomos_quantidade][2] = 33.0;
	electron_z[atomos_quantidade][2] = 200.0;

	electron_y[atomos_quantidade][3] = 300.0;
	electron_z[atomos_quantidade][3] = 1.0;

	atomos_quantidade++;
}

void conecta_eletrons(string procura_amino) {
	connect_electron("N1", "C1");
	connect_electron("C1", "C2");
	connect_electron("C2", "O1");

	if (procura_amino == "ILE") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C3", "C5");
		connect_electron("C5", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C6");
		connect_electron("H10", "C6");
		connect_electron("H11", "C6");
	} else if (procura_amino == "GLU") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("O2", "C5");
		connect_electron("O3", "C5");
	} else if (procura_amino == "ALA") {
		connect_electron("C1", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C3");
	} else if (procura_amino == "ARG") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("N2", "C5");
		connect_electron("C6", "N2");
		connect_electron("N3", "C6");
		connect_electron("N4", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "N2");
		connect_electron("H10", "N3");
		connect_electron("H11", "N3");
		connect_electron("H12", "N4");
		connect_electron("H13", "N4");
	} else if (procura_amino == "LYS") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("C5", "C6");
		connect_electron("N2", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C6");
		connect_electron("H10", "C6");
		connect_electron("H11", "N2");
		connect_electron("H12", "N2");
		connect_electron("H13", "N2");
	} else if (procura_amino == "ASP") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("O2", "C4");
		connect_electron("O3", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
	} else if (procura_amino == "TYR") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("C6", "C5");
		connect_electron("C7", "C6");
		connect_electron("C8", "C7");
		connect_electron("C9", "C8");
		connect_electron("C9", "C4");
		connect_electron("O2", "C7");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C9");
		connect_electron("H6", "C8");
		connect_electron("H7", "O2");
		connect_electron("H8", "C6");
		connect_electron("H9", "C5");
	} else if (procura_amino == "TRP") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("C6", "C4");
		connect_electron("C7", "C6");
		connect_electron("C8", "C7");
		connect_electron("C9", "C8");
		connect_electron("C10", "C9");
		connect_electron("C11", "C10");
		connect_electron("C11", "C6");
		connect_electron("N2", "C5");
		connect_electron("N2", "C11");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C5");
		connect_electron("H6", "N2");
		connect_electron("H7", "C7");
		connect_electron("H8", "C8");
		connect_electron("H9", "C9");
		connect_electron("H10", "C10");
	} else if (procura_amino == "SER") {
		connect_electron("C1", "C3");
		connect_electron("O2", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
	} else if (procura_amino == "THR") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("O2", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
	} else if (procura_amino == "GLY") {
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C1");
	} else if (procura_amino == "MET") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("S1", "C4");
		connect_electron("C5", "S1");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C5");
	} else if (procura_amino == "PHE") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("C6", "C5");
		connect_electron("C7", "C6");
		connect_electron("C8", "C7");
		connect_electron("C9", "C8");
		connect_electron("C9", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C5");
		connect_electron("H6", "C6");
		connect_electron("H7", "C7");
		connect_electron("H8", "C8");
		connect_electron("H9", "C9");
	} else if (procura_amino == "LEU") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("C4", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C5");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C6");
		connect_electron("H10", "C6");
		connect_electron("H11", "C6");
	} else if (procura_amino == "VAL") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C3", "C5");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C5");
	} else if (procura_amino == "ASN") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("N2", "C4");
		connect_electron("O2", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "N2");
		connect_electron("H6", "N2");
	} else if (procura_amino == "GLN") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("N2", "C5");
		connect_electron("O2", "C5");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "N2");
		connect_electron("H8", "N2");
	} else if (procura_amino == "CYS") {
		connect_electron("C3", "C1");
		connect_electron("S1", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
	} else if (procura_amino == "HIS") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("N2", "C5");
		connect_electron("C6", "N2");
		connect_electron("N3", "C6");
		connect_electron("N3", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C5");
		connect_electron("H6", "C6");
		connect_electron("H7", "N3");
	} else if (procura_amino == "PRO") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("N1", "C5");
		connect_electron("H1", "C1");
		connect_electron("H2", "C3");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C5");
		connect_electron("H7", "C5");
	}
}

void read_pdb_amino(string procura_amino) {
	cont_amin = 0;
	printf("Lendo arquivo PDB dos aminoacidos\n");
	string line = "";
	ifstream myfile("files/aminoacidos.pdb");
	if (myfile.is_open()) {
		string tipo;
		string nome_amino;
		string atomo_sigla;
		string atomo_label;
		GLfloat fix_x = 0;
		GLfloat fix_y = 0;
		GLfloat fix_z = 0;
		while (getline(myfile, line)) {
			tipo = line.substr(0, 6);
			if (tipo.erase(tipo.find_last_not_of(" ") + 1) == "ATOM" || tipo == "HETATM") {
				nome_amino = line.substr(17, 3);

				if (nome_amino == procura_amino) {
					cont_amin++;
					atomo_label = line.substr(77, 3);
					atomo_label = atomo_label.erase(atomo_label.find_last_not_of(" ") + 1);

					atomo_sigla = line.substr(77, 1);
					pdbamin_coordenadas_x.resize(cont_amin);
					pdbamin_coordenadas_y.resize(cont_amin);
					pdbamin_coordenadas_z.resize(cont_amin);

					pdbamin_coordenadas_x[cont_amin - 1] = strtof(line.substr(30, 8).c_str(), NULL) - fix_x;
					pdbamin_coordenadas_y[cont_amin - 1] = strtof(line.substr(38, 8).c_str(), NULL) - fix_y;
					pdbamin_coordenadas_z[cont_amin - 1] = strtof(line.substr(46, 8).c_str(), NULL) - fix_z;

					if (cont_amin == 1) {
						printf("Zerando coordenadas\n");
						fix_x = pdbamin_coordenadas_x[cont_amin - 1];
						fix_y = pdbamin_coordenadas_y[cont_amin - 1];
						fix_z = pdbamin_coordenadas_z[cont_amin - 1];
						pdbamin_coordenadas_x[cont_amin - 1] = 0.0;
						pdbamin_coordenadas_y[cont_amin - 1] = 0.0;
						pdbamin_coordenadas_z[cont_amin - 1] = 0.0;
					}
//					printf("%s %s %s %f %f %f\n", atomo_label.c_str(), atomo_sigla.c_str(), nome_amino.c_str(), pdbamin_coordenadas_x[cont_amin - 1], pdbamin_coordenadas_y[cont_amin - 1], pdbamin_coordenadas_z[cont_amin - 1]);
					add_atoml(atomo_sigla);
					mapa_atomo[atomo_label] = atomos_quantidade - 1;
					posx[mapa_atomo[atomo_label]] = pdbamin_coordenadas_x[cont_amin - 1];
					posy[mapa_atomo[atomo_label]] = pdbamin_coordenadas_y[cont_amin - 1];
					posz[mapa_atomo[atomo_label]] = pdbamin_coordenadas_z[cont_amin - 1];
					amino[mapa_atomo[atomo_label]] = contador_amino;
					amino[mapa_atomo[atomo_label]] = get_amino_number(nome_amino.c_str());
					amino_sequencial[mapa_atomo[atomo_label]] = contador_amino;
					atomo_letraN[mapa_atomo[atomo_label]] = get_atom_number(atomo_label.c_str());
//					atomo_letraL[mapa_atomo[atomo_label]] = atomo_label;
//					printf("P: %d %s\n",mapa_atomo[atomo_label], atomo_label.c_str());

					if (atomo_label == "N1" || atomo_label == "C1" || atomo_label == "C2") {
						atomo_base[mapa_atomo[atomo_label]] = true;
					}
					if (atomo_label == "N1" && contador_amino > 0) {
						electron_arested[mapa_atomo[atomo_label]][0][0] = chain_ultimo_atomo;
						electron_arested[mapa_atomo[atomo_label]][0][1] = 1;
						electron_arested[chain_ultimo_atomo][1][0] = mapa_atomo[atomo_label];
						electron_arested[chain_ultimo_atomo][1][1] = 0;
						electron_contador[atomo_label]++;
					} else if (atomo_label == "C2") {
						chain_ultimo_atomo = mapa_atomo[atomo_label];
					}

//					printf("---> %d %d %s %d\n", atomo_letraN[mapa_atomo[atomo_label]], mapa_atomo[atomo_label], atomo_label.c_str(), get_amino_number(nome_amino.c_str()));

				}

//				printf("%s\n",nome_amino.c_str());

			}
		}
//		velocidade_z[mapa_atomo["C1"]] = 5.14;
//		velocidade_z[mapa_atomo["N1"]] = 5.13;
//		velocidade_z[mapa_atomo["C1"]] = 0.04;
//		velocidade_x[mapa_atomo["N1"]] = 0.03;
//		velocidade_z[mapa_atomo["C1"]] = 0.001;
//		velocidade_z[mapa_atomo["N1"]] = 0.003;
		conecta_eletrons(procura_amino);

		myfile.close();
		mapa_atomo.clear();
		electron_contador.clear();
		contador_amino++;
	} else {
		cout << "Unable to open file\n";
	}
}

void add_pdb_amino() {
	for (int i = 0; i < cont_amin; i++) {
		glColor3ub(255, 255, 255);
		glTranslatef(pdbamin_coordenadas_x[i], pdbamin_coordenadas_y[i], pdbamin_coordenadas_z[i]);
		glutSolidCube(0.1f);
//		glutSolidSphere(0.5f, 30, 30);
		glTranslatef(-pdbamin_coordenadas_x[i], -pdbamin_coordenadas_y[i], -pdbamin_coordenadas_z[i]);
	}
}

void load_protein(string PDBID) {
	printf("Lendo proteina a partir do banco: %s\n", PDBID.c_str());
	printf("Calibrando a partir do mysql\n");
	try {
		sql::Driver *driver;
		sql::Connection *con;
		sql::PreparedStatement *pstmt;
		sql::ResultSet *res;
		driver = get_driver_instance();
		con = driver->connect("tcp://127.0.0.1:3306", "a00s_230", "testando");
		con->setSchema("a00s_230");
		// Limpando
		contador_amino = 0;

//		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL HAVING aminoseq IN(1) AND atom IN ('C') ORDER BY i_306344,atomlabel DESC");
//		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL ORDER BY i_306344,atomlabel DESC");
//		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL AND i_306344 IN(1,2) ORDER BY i_306344,atomlabel DESC");
		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL ORDER BY i_306344,atomlabel DESC");
		pstmt->setString(1, PDBID);
		res = pstmt->executeQuery();
		string nome_amino;
		string atomo_sigla;
		string atomo_label;
		int aminoseq = -1;
//		contador_amino = -1;
		while (res->next()) {
//			printf("%s %s\n", res->getString("amino").c_str(), res->getString("atomlabel").c_str());
			if (res->getInt("aminoseq") != aminoseq) {
				if (aminoseq != -1) {
					contador_amino++;
					conecta_eletrons(nome_amino);
				}
				aminoseq = res->getInt("aminoseq");
				mapa_atomo.clear();
				electron_contador.clear();
			}
			nome_amino = res->getString("amino");
			aminoseq = res->getInt("aminoseq");
			cont_amin++;
			atomo_label = res->getString("atomlabel");
			atomo_sigla = res->getString("atom");
			pdbamin_coordenadas_x.resize(cont_amin);
			pdbamin_coordenadas_y.resize(cont_amin);
			pdbamin_coordenadas_z.resize(cont_amin);
			pdbamin_coordenadas_x[cont_amin - 1] = strtof(res->getString("x").c_str(), NULL);
			pdbamin_coordenadas_y[cont_amin - 1] = strtof(res->getString("y").c_str(), NULL);
			pdbamin_coordenadas_z[cont_amin - 1] = strtof(res->getString("z").c_str(), NULL);
			add_atoml(atomo_sigla);
			mapa_atomo[atomo_label] = atomos_quantidade - 1;
			posx[mapa_atomo[atomo_label]] = pdbamin_coordenadas_x[cont_amin - 1];
			posy[mapa_atomo[atomo_label]] = pdbamin_coordenadas_y[cont_amin - 1];
			posz[mapa_atomo[atomo_label]] = pdbamin_coordenadas_z[cont_amin - 1];
			amino[mapa_atomo[atomo_label]] = contador_amino;
			amino[mapa_atomo[atomo_label]] = get_amino_number(nome_amino.c_str());
			amino_sequencial[mapa_atomo[atomo_label]] = contador_amino;
			atomo_letraN[mapa_atomo[atomo_label]] = get_atom_number(atomo_label.c_str());
			atomo_letraL[mapa_atomo[atomo_label]] = atomo_label;
//			printf("P: %d %s\n", mapa_atomo[atomo_label], atomo_label.c_str());

			if (atomo_label == "N1" || atomo_label == "C1" || atomo_label == "C2") {
				atomo_base[mapa_atomo[atomo_label]] = true;
				velocidade_z[mapa_atomo["C1"]] = 0.04;
				velocidade_x[mapa_atomo["N1"]] = 0.08;
			}
			if (atomo_label == "N1" && contador_amino > 0) {
//				printf("Interligando com o ultimo atomo %d\n",chain_ultimo_atomo);
				electron_arested[mapa_atomo[atomo_label]][0][0] = chain_ultimo_atomo;
				electron_arested[mapa_atomo[atomo_label]][0][1] = 1;
				electron_arested[chain_ultimo_atomo][1][0] = mapa_atomo[atomo_label];
				electron_arested[chain_ultimo_atomo][1][1] = 0;
				electron_contador[atomo_label]++;
			} else if (atomo_label == "C2") {
//				printf("Setando ultimo atomo %d\n",mapa_atomo[atomo_label]);
				chain_ultimo_atomo = mapa_atomo[atomo_label];
			}
		}
//		velocidade_z[mapa_atomo["C1"]] = 0.01;
//		velocidade_x[mapa_atomo["N1"]] = 0.03;
		conecta_eletrons(nome_amino);
		delete res;
		delete pstmt;
		delete con;
		printf("PDBid Loaded\n");
	} catch (sql::SQLException &e) {
		printf("%d\n", e.getErrorCode());
	}
}

GLfloat compare_protein_calculate() {
//	printf("------- CPC-------------\n");
	uint tamanho_vetor_1 = AminoTemp1.size();
	uint tamanho_vetor_2 = AminoTemp2.size();
	GLfloat distance_local = 0.0;
	GLfloat distance_local2 = 0.0;
	for (uint i = 0; i < tamanho_vetor_1; ++i) {
		distance_local = 0.0;
		for (uint ii = 0; ii < tamanho_vetor_2; ++ii) {
			distance_local += abs(sqrt(pow((AminoTemp2[ii].x - AminoTemp1[i].x), 2.0) + pow((AminoTemp2[ii].y - AminoTemp1[i].y), 2.0) + pow((AminoTemp2[ii].z - AminoTemp1[i].z), 2.0)));
		}
		distance_local2 += distance_local;
//		printf("C %d %s\n", AminoTemp1[i].aminoseq, AminoTemp1[i].label.c_str());
//		if (AminoTemp1[i].label == "O1") {
//			printf("------->O1 seq %d \n", AminoTemp1[i].aminoseq);
//		}
		if (atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MIN"] == 0) {
//			if (AminoTemp1[i].label == "O1") {
//				printf("P1 %f %d %d\n",distance_local, tamanho_vetor_1, AminoTemp1[i].aminoseq);
//			}
			atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MIN"] = distance_local;
			atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MAX"] = distance_local;
		} else {
//			if (AminoTemp1[i].label == "O1") {
//				printf("P2 %f %d %d (%f %f)\n",distance_local, tamanho_vetor_1, AminoTemp1[i].aminoseq, atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MIN"],atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MAX"]);
//			}
			if (distance_local < atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MIN"]) {
				atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MIN"] = distance_local;
				if (AminoTemp1[i].label == "O1") {
//					printf("P3 %f\n",distance_local);
				}
			}
			if (distance_local > atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MAX"]) {
				atom_statistic[AminoTemp1[i].aminoseq][AminoTemp1[i].label]["MAX"] = distance_local;
				if (AminoTemp1[i].label == "O1") {
//					printf("P4 %f\n",distance_local);
				}
			}
		}
	}
	return distance_local2;
}

GLfloat compare_protein_calculateMD() {
	uint tamanho_vetor_1 = AminoTemp1MD.size();
	uint tamanho_vetor_2 = AminoTemp2MD.size();
	GLfloat distance_local = 0.0;
	GLfloat distance_local2 = 0.0;
	for (uint i = 0; i < tamanho_vetor_1; ++i) {
		distance_local = 0.0;
		for (uint ii = 0; ii < tamanho_vetor_2; ++ii) {
			distance_local += abs(sqrt(pow((AminoTemp2MD[ii].x - AminoTemp1MD[i].x), 2.0) + pow((AminoTemp2MD[ii].y - AminoTemp1MD[i].y), 2.0) + pow((AminoTemp2MD[ii].z - AminoTemp1MD[i].z), 2.0)));
		}
		distance_local2 += distance_local;
//		printf("SequencialB %d %s\n",AminoTemp1MD[i].aminoseq,AminoTemp1MD[i].label.c_str());
		atom_statistic[AminoTemp1MD[i].aminoseq][AminoTemp1MD[i].label]["MD"] = distance_local;
	}
	return distance_local2;
}

void compare_protein_build_MD(int c_distancia) {
	uint tamanho_vetor = pdb_analise.size();
	int last_seq = -1;
	int cont_seq = 0;
	int cont_vector = -1;
	int i1c = 0;
	int i2c = c_distancia;
	GLfloat soma_total = 0.0;
//	printf("Comparando\n");
	for (GLint i = 0; i < atomos_quantidade; i++) {
//		printf("3d\n");
		if (last_seq > 0) {
			if (last_seq != amino_sequencial[i]) {
				cont_seq++;
				cont_vector = -1;
				if (cont_seq > i2c) {
					soma_total += compare_protein_calculateMD();
					cont_seq = 1; // antes tava 0
					i1c++;
					i2c++;
					i = 0;
				}
			}
		} else {
			last_seq = amino_sequencial[i];
//			printf("LS %d\n", amino_sequencial[i]);
		}
//		printf("Last seq: %d\n", last_seq);
		if (i1c == cont_seq) {
			cont_vector++;
			AminoTemp1MD.resize(cont_vector + 1);
			AminoTemp1MD[cont_vector].x = posx[i];
			AminoTemp1MD[cont_vector].y = posy[i];
			AminoTemp1MD[cont_vector].z = posz[i];
			AminoTemp1MD[cont_vector].aminoseq = amino_sequencial[i];
			AminoTemp1MD[cont_vector].label = atomo_letraL[i].c_str();
//			printf("Sequencial %d   Label %s  \n",amino_sequencial[i], atomo_letraL[i].c_str());
		}
		if (i2c == cont_seq) {
			if (i2c != i1c) {
				cont_vector++;
			}
			AminoTemp2MD.resize(cont_vector + 1);
			AminoTemp2MD[cont_vector].x = posx[i];
			AminoTemp2MD[cont_vector].y = posy[i];
			AminoTemp2MD[cont_vector].z = posz[i];
		}
		last_seq = amino_sequencial[i];
	}
	soma_total += compare_protein_calculateMD();
//	printf("Total final1: %f\n", soma_total);
}

GLfloat compare_protein_build(int c_distancia) {
//	c_distancia++;
	uint tamanho_vetor = pdb_analise.size();
//	printf("Tamanho vetor %d\n", tamanho_vetor);
	int last_seq = -1;
	int cont_seq = 0;
	int cont_vector = -1;
	int i1c = 0;
	int i2c = c_distancia;
	GLfloat soma_total = 0.0;
	for (uint i = 0; i < tamanho_vetor; ++i) {
//		printf("Aminoseq %d\n",pdb_analise[i].aminoseq);
		if (last_seq > 1) {
			if (last_seq != pdb_analise[i].aminoseq) {
				cont_seq++;
				cont_vector = -1;
				if (cont_seq > i2c) {
//					printf("Chama cpc 1\n");
					soma_total += compare_protein_calculate();
					cont_seq = 1;
					i1c++;
					i2c++;
					i = 0;
				}
			}
		} else {
			last_seq = pdb_analise[i].aminoseq;
		}
//		printf("ContsetA %d\n",cont_seq);
		if (i1c == cont_seq) {
//			printf("Aqui 1: %d %d %s\n", cont_vector, pdb_analise[i].aminoseq - 1, pdb_analise[i].label.c_str());
			cont_vector++;
			AminoTemp1.resize(cont_vector + 1);
			AminoTemp1[cont_vector].x = pdb_analise[i].x;
			AminoTemp1[cont_vector].y = pdb_analise[i].y;
			AminoTemp1[cont_vector].z = pdb_analise[i].z;
			AminoTemp1[cont_vector].aminoseq = pdb_analise[i].aminoseq - 1;
			AminoTemp1[cont_vector].label = pdb_analise[i].label;
		}
//		printf("ContsetB %d\n",cont_seq);
		if (i2c == cont_seq) {
//			printf("Aqui 2\n");
			if (i2c != i1c) {
				cont_vector++;
			}
			AminoTemp2.resize(cont_vector + 1);
			AminoTemp2[cont_vector].x = pdb_analise[i].x;
			AminoTemp2[cont_vector].y = pdb_analise[i].y;
			AminoTemp2[cont_vector].z = pdb_analise[i].z;
		}
//		printf("ContsetC %d\n",cont_seq);
		last_seq = pdb_analise[i].aminoseq;
	}
//	printf("Chama cpc 2\n");
	soma_total += compare_protein_calculate();
	return (soma_total);
//	printf("Total final2: %f\n", soma_total);
}

GLfloat load_protein_position(string PDBID, string model, int distance) {
//	printf("Lendo proteina para analise: %s %s\n", PDBID.c_str(), model.c_str());
	try {
		sql::Driver *driver2;
		sql::Connection *con2;
		sql::PreparedStatement *pstmt2;
		sql::ResultSet *res2;
		driver2 = get_driver_instance();
		con2 = driver2->connect("tcp://127.0.0.1:3306", "a00s_230", "testando");
		con2->setSchema("a00s_230");
//		pstmt = con->prepareStatement("SELECT i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL ORDER BY i_306344,atomlabel DESC");
//		string sql_final = "SELECT i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_331770 atomlabel FROM a_306280 WHERE i_307676='" + PDBID + "' AND i_306401='" + model + "' AND i_331770 IS NOT NULL AND i_306344 IN(1,2) ORDER BY i_306344,atomlabel DESC";
		string sql_final = "SELECT i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_331770 atomlabel FROM a_306280 WHERE i_307676='" + PDBID + "' AND i_306401='" + model + "' AND i_331770 IS NOT NULL ORDER BY i_306344,atomlabel DESC";
		pstmt2 = con2->prepareStatement(sql_final);
//		pstmt2->setString(1, PDBID);
//		pstmt2->setInt(1, model);
		res2 = pstmt2->executeQuery();
		int cont_atom = -1;
		while (res2->next()) {
			cont_atom++;
//			printf("x1:\n");
//			printf("x: %f\n",res->getDouble("x"));
			pdb_analise.resize(cont_atom + 1);
			pdb_analise[cont_atom].aminoseq = res2->getInt("aminoseq");
			pdb_analise[cont_atom].label = res2->getString("atomlabel").c_str();
			pdb_analise[cont_atom].x = res2->getDouble("x");
			pdb_analise[cont_atom].y = res2->getDouble("y");
			pdb_analise[cont_atom].z = res2->getDouble("z");
//			if(pdb_analise[cont_atom].label == "O1"){
//				printf("O1: %f %f %f\n",pdb_analise[cont_atom].x,pdb_analise[cont_atom].y,pdb_analise[cont_atom].z);
//			}
		}
		delete res2;
		delete pstmt2;
		delete con2;
//		printf("PDBid Loaded\n");
//		compare_protein_build_MD(0);
		return (compare_protein_build(distance));
	} catch (sql::SQLException &e) {
		printf("%d\n", e.getErrorCode());
	}
	return 0.0;
}

void load_protein_models(string PDBID) {
	printf("Vendo quantidade de modelos: %s\n", PDBID.c_str());
	try {
//		map< int, map< string, map< string, GLfloat > > > mapa_teste;
//		mapa_teste[1]["C1"]["MIN"] = 10.1;
//		printf("Res: %f\n",mapa_teste[1]["C1"]["MIN"]);

		GLfloat calculated_pdb_min = 0;
		GLfloat calculated_pdb_max = 0;
		sql::Driver *driver;
		sql::Connection *con;
		sql::PreparedStatement *pstmt;
		sql::ResultSet *res;
		driver = get_driver_instance();
		con = driver->connect("tcp://127.0.0.1:3306", "a00s_230", "testando");
		con->setSchema("a00s_230");
		pstmt = con->prepareStatement("SELECT i_306401 FROM a_306280 WHERE i_307676=? GROUP BY i_306401");
		pstmt->setString(1, PDBID);
		res = pstmt->executeQuery();
		GLfloat resultado_local = 0;
		while (res->next()) {
//			printf("Modelo %d\n",res->getInt(1));
			resultado_local = load_protein_position(PDBID, res->getString(1), 1);
			if (calculated_pdb_min == 0) {
				calculated_pdb_min = resultado_local;
				calculated_pdb_max = resultado_local;
			} else {
				if (resultado_local < calculated_pdb_min) {
					calculated_pdb_min = resultado_local;
				}
				if (resultado_local > calculated_pdb_max) {
					calculated_pdb_max = resultado_local;
				}
			}
		}
		delete res;
		delete pstmt;
		delete con;
		printf("MINMAX %f %f\n", calculated_pdb_min, calculated_pdb_max);
	} catch (sql::SQLException &e) {
		printf("%d\n", e.getErrorCode());
	}
}
