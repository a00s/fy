#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include "mysql_connection.h"
#include <cppconn/driver.h>
#include <cppconn/exception.h>
#include <cppconn/resultset.h>
#include <cppconn/statement.h>
#include <cppconn/prepared_statement.h>
using namespace std;

vector<GLfloat> pdbamin_coordenadas_x;
vector<GLfloat> pdbamin_coordenadas_y;
vector<GLfloat> pdbamin_coordenadas_z;

struct VectorProtein {
	GLint aminoseq;
	string label;
	GLfloat x;
	GLfloat y;
	GLfloat z;
};
vector<VectorProtein> pdb_analise;

struct StructAmino {
	GLfloat x;
	GLfloat y;
	GLfloat z;
};
vector<StructAmino> AminoTemp1;
vector<StructAmino> AminoTemp2;

vector<StructAmino> AminoTemp1MD;
vector<StructAmino> AminoTemp2MD;

GLint cont_amin = 0;
extern int atomos_quantidade;
extern char atomo_letra[1000];
extern GLfloat nucleo_proximity[1000];
extern GLfloat nucleo_proximity_free[1000];
extern GLfloat nucleo_proximity_HB[1000];
extern GLint electron_quantidade[1000];
extern GLfloat electron_raio[1000];
extern GLfloat electron_raio_HB[1000];
extern GLint electron_arested[1000][4][2];
extern GLfloat massa[1000];
extern GLfloat posx[1000];
extern GLfloat posy[1000];
extern GLfloat posz[1000];
extern GLfloat velocidade_x[1000]; // velocidade
extern GLfloat velocidade_y[1000]; // velocidade
extern GLfloat velocidade_z[1000]; // velocidade
extern GLfloat electron_y[1000][4];
extern GLfloat electron_z[1000][4];
extern GLint amino[1000];
extern GLint amino_sequencial[1000];
extern GLint atomo_letraN[1000];
extern GLint contador_amino;
extern int get_amino_number(const char *amino_sigla);
extern int get_atom_number(const char *atomo_letra_local);
extern bool atomo_base[1000];
extern GLint chain_ultimo_atomo;
map<string, int> mapa_atomo;
map<string, int> electron_contador;

void connect_electron(string atomo1, string atomo2) {
	electron_arested[mapa_atomo[atomo1]][electron_contador[atomo1]][0] = mapa_atomo[atomo2];
	electron_arested[mapa_atomo[atomo1]][electron_contador[atomo1]][1] = electron_contador[atomo2];
	electron_arested[mapa_atomo[atomo2]][electron_contador[atomo2]][0] = mapa_atomo[atomo1];
	electron_arested[mapa_atomo[atomo2]][electron_contador[atomo2]][1] = electron_contador[atomo1];
	electron_contador[atomo1]++;
	electron_contador[atomo2]++;
}

void add_atoml(string atomo_ll) {
	if (atomos_quantidade == 998) {
		printf("Quantidade de atomos chegou no limite \n");
		return;
	}

	if (atomo_ll == "C") {
		atomo_letra[atomos_quantidade] = 'C';
		nucleo_proximity[atomos_quantidade] = 1.7;
		nucleo_proximity_free[atomos_quantidade] = 3.4; // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 4;
		electron_raio[atomos_quantidade] = 3.4;
		massa[atomos_quantidade] = 12.0107;
	} else if (atomo_ll == "H") {
		atomo_letra[atomos_quantidade] = 'H';
		nucleo_proximity[atomos_quantidade] = 1.2;
		nucleo_proximity_free[atomos_quantidade] = 2.4;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 1;
		electron_raio[atomos_quantidade] = 2.4;
		electron_raio_HB[atomos_quantidade] = 2.4;
		massa[atomos_quantidade] = 1.0079;
		nucleo_proximity_HB[atomos_quantidade] = 1.2;

//		electron_y_HB[atomos_quantidade] = 270;
	} else if (atomo_ll == "O") {
		atomo_letra[atomos_quantidade] = 'O';
		nucleo_proximity[atomos_quantidade] = 1.52;
		nucleo_proximity_free[atomos_quantidade] = 3.04;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 2;
		electron_raio[atomos_quantidade] = 3.04;
		electron_raio_HB[atomos_quantidade] = 3.04;
		massa[atomos_quantidade] = 15.099;
		nucleo_proximity_HB[atomos_quantidade] = 1.52;

//		electron_y_HB[atomos_quantidade] = 90;
	} else if (atomo_ll == "N") {
		atomo_letra[atomos_quantidade] = 'N';
//		nucleo_proximity[atomos_quantidade] = 1.55; // original
		nucleo_proximity[atomos_quantidade] = 1.46; // average collected
		nucleo_proximity_free[atomos_quantidade] = 3.1;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 4;
		electron_raio[atomos_quantidade] = 2.92;
		electron_raio_HB[atomos_quantidade] = 3.1;
		massa[atomos_quantidade] = 14.0067;
		nucleo_proximity_HB[atomos_quantidade] = 1.55;
	} else if (atomo_ll == "S") {
		atomo_letra[atomos_quantidade] = 'S';
		nucleo_proximity[atomos_quantidade] = 1.8;
		nucleo_proximity_free[atomos_quantidade] = 3.6;  // Van der waals angstron
		electron_quantidade[atomos_quantidade] = 4;
		electron_raio[atomos_quantidade] = 3.6;
		massa[atomos_quantidade] = 32.06;
	}
	posx[atomos_quantidade] = 1.0;
	posy[atomos_quantidade] = 3.0;
	posz[atomos_quantidade] = 2.0;
	velocidade_x[atomos_quantidade] = 0.0;
	velocidade_y[atomos_quantidade] = 0.0;
	velocidade_z[atomos_quantidade] = 0.0;
	electron_y[atomos_quantidade][0] = 345.0;
	electron_z[atomos_quantidade][0] = 350.0;

	electron_y[atomos_quantidade][1] = 100.0;
	electron_z[atomos_quantidade][1] = 150.0;

	electron_y[atomos_quantidade][2] = 33.0;
	electron_z[atomos_quantidade][2] = 200.0;

	electron_y[atomos_quantidade][3] = 300.0;
	electron_z[atomos_quantidade][3] = 1.0;

	atomos_quantidade++;
}

void conecta_eletrons(string procura_amino) {
	connect_electron("N1", "C1");
	connect_electron("C1", "C2");
	connect_electron("C2", "O1");

	if (procura_amino == "ILE") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C3", "C5");
		connect_electron("C5", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C6");
		connect_electron("H10", "C6");
		connect_electron("H11", "C6");
	} else if (procura_amino == "GLU") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
	} else if (procura_amino == "ALA") {
		connect_electron("C1", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C3");
	} else if (procura_amino == "ARG") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("N2", "C5");
		connect_electron("C6", "N2");
		connect_electron("N3", "C6");
		connect_electron("N4", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "N2");
		connect_electron("H10", "N3");
		connect_electron("H11", "N3");
		connect_electron("H12", "N4");
		connect_electron("H13", "N4");
	} else if (procura_amino == "LYS") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("C5", "C6");
		connect_electron("N2", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C6");
		connect_electron("H10", "C6");
		connect_electron("H11", "N2");
		connect_electron("H12", "N2");
		connect_electron("H13", "N2");
	} else if (procura_amino == "ASP") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("O2", "C4");
		connect_electron("O3", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
	} else if (procura_amino == "TYR") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("C6", "C5");
		connect_electron("C7", "C6");
		connect_electron("C8", "C7");
		connect_electron("C9", "C8");
		connect_electron("C9", "C4");
		connect_electron("O2", "C7");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C9");
		connect_electron("H6", "C8");
		connect_electron("H7", "O2");
		connect_electron("H8", "C6");
		connect_electron("H9", "C5");
	} else if (procura_amino == "TRP") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("C6", "C4");
		connect_electron("C7", "C6");
		connect_electron("C8", "C7");
		connect_electron("C9", "C8");
		connect_electron("C10", "C9");
		connect_electron("C11", "C10");
		connect_electron("C11", "C6");
		connect_electron("N2", "C5");
		connect_electron("N2", "C11");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C5");
		connect_electron("H6", "N2");
		connect_electron("H7", "C7");
		connect_electron("H8", "C8");
		connect_electron("H9", "C9");
		connect_electron("H10", "C10");
	} else if (procura_amino == "SER") {
		connect_electron("C1", "C3");
		connect_electron("O2", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
	} else if (procura_amino == "THR") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("O2", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
	} else if (procura_amino == "GLY") {
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C1");
	} else if (procura_amino == "MET") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("S1", "C4");
		connect_electron("C5", "S1");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C5");
	} else if (procura_amino == "PHE") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("C6", "C5");
		connect_electron("C7", "C6");
		connect_electron("C8", "C7");
		connect_electron("C9", "C8");
		connect_electron("C9", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C5");
		connect_electron("H6", "C6");
		connect_electron("H7", "C7");
		connect_electron("H8", "C8");
		connect_electron("H9", "C9");
	} else if (procura_amino == "LEU") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C4", "C5");
		connect_electron("C4", "C6");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C5");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C6");
		connect_electron("H10", "C6");
		connect_electron("H11", "C6");
	} else if (procura_amino == "VAL") {
		connect_electron("C1", "C3");
		connect_electron("C3", "C4");
		connect_electron("C3", "C5");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "C5");
		connect_electron("H8", "C5");
		connect_electron("H9", "C5");
	} else if (procura_amino == "ASN") {
		connect_electron("C1", "C3");
		connect_electron("C4", "C3");
		connect_electron("N2", "C4");
		connect_electron("O2", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "N2");
		connect_electron("H6", "N2");
	} else if (procura_amino == "GLN") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("N2", "C5");
		connect_electron("O2", "C5");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C4");
		connect_electron("H6", "C4");
		connect_electron("H7", "N2");
		connect_electron("H8", "N2");
	} else if (procura_amino == "CYS") {
		connect_electron("C3", "C1");
		connect_electron("S1", "C3");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
	} else if (procura_amino == "HIS") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("N2", "C5");
		connect_electron("C6", "N2");
		connect_electron("N3", "C6");
		connect_electron("N3", "C4");
		connect_electron("H1", "N1");
		connect_electron("H2", "C1");
		connect_electron("H3", "C3");
		connect_electron("H4", "C3");
		connect_electron("H5", "C5");
		connect_electron("H6", "C6");
		connect_electron("H7", "N3");
	} else if (procura_amino == "PRO") {
		connect_electron("C3", "C1");
		connect_electron("C4", "C3");
		connect_electron("C5", "C4");
		connect_electron("N1", "C5");
		connect_electron("H1", "C1");
		connect_electron("H2", "C3");
		connect_electron("H3", "C3");
		connect_electron("H4", "C4");
		connect_electron("H5", "C4");
		connect_electron("H6", "C5");
		connect_electron("H7", "C5");
	}
}

void read_pdb_amino(string procura_amino) {
	cont_amin = 0;
	printf("Lendo arquivo PDB dos aminoacidos\n");
	string line = "";
	ifstream myfile("files/aminoacidos.pdb");
	if (myfile.is_open()) {
		string tipo;
		string nome_amino;
		string atomo_sigla;
		string atomo_label;
		GLfloat fix_x = 0;
		GLfloat fix_y = 0;
		GLfloat fix_z = 0;
		while (getline(myfile, line)) {
			tipo = line.substr(0, 6);
			if (tipo.erase(tipo.find_last_not_of(" ") + 1) == "ATOM" || tipo == "HETATM") {
				nome_amino = line.substr(17, 3);

				if (nome_amino == procura_amino) {
					cont_amin++;
					atomo_label = line.substr(77, 3);
					atomo_label = atomo_label.erase(atomo_label.find_last_not_of(" ") + 1);

					atomo_sigla = line.substr(77, 1);
					pdbamin_coordenadas_x.resize(cont_amin);
					pdbamin_coordenadas_y.resize(cont_amin);
					pdbamin_coordenadas_z.resize(cont_amin);

					pdbamin_coordenadas_x[cont_amin - 1] = strtof(line.substr(30, 8).c_str(), NULL) - fix_x;
					pdbamin_coordenadas_y[cont_amin - 1] = strtof(line.substr(38, 8).c_str(), NULL) - fix_y;
					pdbamin_coordenadas_z[cont_amin - 1] = strtof(line.substr(46, 8).c_str(), NULL) - fix_z;

					if (cont_amin == 1) {
						printf("Zerando coordenadas\n");
						fix_x = pdbamin_coordenadas_x[cont_amin - 1];
						fix_y = pdbamin_coordenadas_y[cont_amin - 1];
						fix_z = pdbamin_coordenadas_z[cont_amin - 1];
						pdbamin_coordenadas_x[cont_amin - 1] = 0.0;
						pdbamin_coordenadas_y[cont_amin - 1] = 0.0;
						pdbamin_coordenadas_z[cont_amin - 1] = 0.0;
					}
//					printf("%s %s %s %f %f %f\n", atomo_label.c_str(), atomo_sigla.c_str(), nome_amino.c_str(), pdbamin_coordenadas_x[cont_amin - 1], pdbamin_coordenadas_y[cont_amin - 1], pdbamin_coordenadas_z[cont_amin - 1]);
					add_atoml(atomo_sigla);
					mapa_atomo[atomo_label] = atomos_quantidade - 1;
					posx[mapa_atomo[atomo_label]] = pdbamin_coordenadas_x[cont_amin - 1];
					posy[mapa_atomo[atomo_label]] = pdbamin_coordenadas_y[cont_amin - 1];
					posz[mapa_atomo[atomo_label]] = pdbamin_coordenadas_z[cont_amin - 1];
					amino[mapa_atomo[atomo_label]] = contador_amino;
					amino[mapa_atomo[atomo_label]] = get_amino_number(nome_amino.c_str());
					amino_sequencial[mapa_atomo[atomo_label]] = contador_amino;
					atomo_letraN[mapa_atomo[atomo_label]] = get_atom_number(atomo_label.c_str());
					if (atomo_label == "N1" || atomo_label == "C1" || atomo_label == "C2") {
						atomo_base[mapa_atomo[atomo_label]] = true;
					}
					if (atomo_label == "N1" && contador_amino > 0) {
						electron_arested[mapa_atomo[atomo_label]][0][0] = chain_ultimo_atomo;
						electron_arested[mapa_atomo[atomo_label]][0][1] = 1;
						electron_arested[chain_ultimo_atomo][1][0] = mapa_atomo[atomo_label];
						electron_arested[chain_ultimo_atomo][1][1] = 0;
						electron_contador[atomo_label]++;
					} else if (atomo_label == "C2") {
						chain_ultimo_atomo = mapa_atomo[atomo_label];
					}

//					printf("---> %d %d %s %d\n", atomo_letraN[mapa_atomo[atomo_label]], mapa_atomo[atomo_label], atomo_label.c_str(), get_amino_number(nome_amino.c_str()));

				}

//				printf("%s\n",nome_amino.c_str());

			}
		}
//		velocidade_z[mapa_atomo["C1"]] = 0.14;
//		velocidade_z[mapa_atomo["N1"]] = 0.13;
//		velocidade_z[mapa_atomo["C1"]] = 0.04;
//		velocidade_z[mapa_atomo["N1"]] = 0.03;
		velocidade_z[mapa_atomo["C1"]] = 0.001;
		velocidade_z[mapa_atomo["N1"]] = 0.003;
		conecta_eletrons(procura_amino);

		myfile.close();
		mapa_atomo.clear();
		electron_contador.clear();
		contador_amino++;
	} else {
		cout << "Unable to open file\n";
	}
}

void add_pdb_amino() {
	for (int i = 0; i < cont_amin; i++) {
		glColor3ub(255, 255, 255);
		glTranslatef(pdbamin_coordenadas_x[i], pdbamin_coordenadas_y[i], pdbamin_coordenadas_z[i]);
		glutSolidCube(0.1f);
//		glutSolidSphere(0.5f, 30, 30);
		glTranslatef(-pdbamin_coordenadas_x[i], -pdbamin_coordenadas_y[i], -pdbamin_coordenadas_z[i]);
	}
}

void load_protein(string PDBID) {
	printf("Lendo proteina a partir do banco: %s\n", PDBID.c_str());
	printf("Calibrando a partir do mysql\n");
	try {
		sql::Driver *driver;
		sql::Connection *con;
		sql::PreparedStatement *pstmt;
		sql::ResultSet *res;
		driver = get_driver_instance();
		con = driver->connect("tcp://127.0.0.1:3306", "a00s_230", "testando");
		con->setSchema("a00s_230");

//		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL HAVING aminoseq IN(1) AND atom IN ('C') ORDER BY i_306344,atomlabel DESC");
//		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL ORDER BY i_306344,atomlabel DESC");
		pstmt = con->prepareStatement("SELECT i_306337 amino, i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_306408 atom, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL AND i_306344=1 ORDER BY i_306344,atomlabel DESC");
		pstmt->setString(1, PDBID);
		res = pstmt->executeQuery();
		string nome_amino;
		string atomo_sigla;
		string atomo_label;
		int aminoseq = -1;
//		contador_amino = -1;
		while (res->next()) {
//			printf("%s %s\n", res->getString("amino").c_str(), res->getString("atomlabel").c_str());
			if (res->getInt("aminoseq") != aminoseq) {
				if (aminoseq != -1) {
					contador_amino++;
					conecta_eletrons(nome_amino);
				}
				aminoseq = res->getInt("aminoseq");
				mapa_atomo.clear();
				electron_contador.clear();
			}
			nome_amino = res->getString("amino");
			aminoseq = res->getInt("aminoseq");
			cont_amin++;
			atomo_label = res->getString("atomlabel");
			atomo_sigla = res->getString("atom");
			pdbamin_coordenadas_x.resize(cont_amin);
			pdbamin_coordenadas_y.resize(cont_amin);
			pdbamin_coordenadas_z.resize(cont_amin);
			pdbamin_coordenadas_x[cont_amin - 1] = strtof(res->getString("x").c_str(), NULL);
			pdbamin_coordenadas_y[cont_amin - 1] = strtof(res->getString("y").c_str(), NULL);
			pdbamin_coordenadas_z[cont_amin - 1] = strtof(res->getString("z").c_str(), NULL);
			add_atoml(atomo_sigla);
			mapa_atomo[atomo_label] = atomos_quantidade - 1;
			posx[mapa_atomo[atomo_label]] = pdbamin_coordenadas_x[cont_amin - 1];
			posy[mapa_atomo[atomo_label]] = pdbamin_coordenadas_y[cont_amin - 1];
			posz[mapa_atomo[atomo_label]] = pdbamin_coordenadas_z[cont_amin - 1];
			amino[mapa_atomo[atomo_label]] = contador_amino;
			amino[mapa_atomo[atomo_label]] = get_amino_number(nome_amino.c_str());
			amino_sequencial[mapa_atomo[atomo_label]] = contador_amino;
			atomo_letraN[mapa_atomo[atomo_label]] = get_atom_number(atomo_label.c_str());
			if (atomo_label == "N1" || atomo_label == "C1" || atomo_label == "C2") {
				atomo_base[mapa_atomo[atomo_label]] = true;
			}
			if (atomo_label == "N1" && contador_amino > 0) {
//				printf("Interligando com o ultimo atomo %d\n",chain_ultimo_atomo);
				electron_arested[mapa_atomo[atomo_label]][0][0] = chain_ultimo_atomo;
				electron_arested[mapa_atomo[atomo_label]][0][1] = 1;
				electron_arested[chain_ultimo_atomo][1][0] = mapa_atomo[atomo_label];
				electron_arested[chain_ultimo_atomo][1][1] = 0;
				electron_contador[atomo_label]++;
			} else if (atomo_label == "C2") {
//				printf("Setando ultimo atomo %d\n",mapa_atomo[atomo_label]);
				chain_ultimo_atomo = mapa_atomo[atomo_label];
			}
		}
//		velocidade_z[mapa_atomo["C1"]] = 0.001;
//		velocidade_z[mapa_atomo["N1"]] = 0.003;
		conecta_eletrons(nome_amino);
		delete res;
		delete pstmt;
		delete con;
		printf("PDBid Loaded\n");
	} catch (sql::SQLException &e) {
		printf("%d\n", e.getErrorCode());
	}
}

GLfloat compare_protein_calculate() {
//	printf("Comparando\n");
	uint tamanho_vetor_1 = AminoTemp1.size();
	uint tamanho_vetor_2 = AminoTemp2.size();
	GLfloat distance_local = 0.0;
	for (uint i = 0; i < tamanho_vetor_1; ++i) {
		for (uint ii = 0; ii < tamanho_vetor_2; ++ii) {
//			printf("%f %f %f / %f %f %f\n",AminoTemp1[i].x,AminoTemp1[i].y,AminoTemp1[i].z,AminoTemp2[ii].x,AminoTemp2[ii].y,AminoTemp2[ii].z);
//			printf("D: (%f %f %f | %f %f %f)   %f        %f\n", AminoTemp2[i].x, AminoTemp2[i].y, AminoTemp2[i].z, AminoTemp2[ii].x, AminoTemp2[ii].y, AminoTemp2[ii].z, abs(sqrt(pow((AminoTemp2[ii].x - AminoTemp1[i].x), 2.0) + pow((AminoTemp2[ii].y - AminoTemp1[i].y), 2.0) + pow((AminoTemp2[ii].z - AminoTemp1[i].z), 2.0))), distance_local);
			distance_local += abs(sqrt(pow((AminoTemp2[ii].x - AminoTemp1[i].x), 2.0) + pow((AminoTemp2[ii].y - AminoTemp1[i].y), 2.0) + pow((AminoTemp2[ii].z - AminoTemp1[i].z), 2.0)));
		}
	}
//	printf("%f\n",distance_local);
	return distance_local;
}

GLfloat compare_protein_calculateMD() {
	uint tamanho_vetor_1 = AminoTemp1MD.size();
	uint tamanho_vetor_2 = AminoTemp2MD.size();
	GLfloat distance_local = 0.0;
	for (uint i = 0; i < tamanho_vetor_1; ++i) {
		for (uint ii = 0; ii < tamanho_vetor_2; ++ii) {
//			printf("%f %f %f / %f %f %f\n",AminoTemp1[i].x,AminoTemp1[i].y,AminoTemp1[i].z,AminoTemp2[ii].x,AminoTemp2[ii].y,AminoTemp2[ii].z);
			distance_local += abs(sqrt(pow((AminoTemp2MD[ii].x - AminoTemp1MD[i].x), 2.0) + pow((AminoTemp2MD[ii].y - AminoTemp1MD[i].y), 2.0) + pow((AminoTemp2MD[ii].z - AminoTemp1MD[i].z), 2.0)));
		}
	}
//	printf("%f\n",distance_local);
	return distance_local;
}

void compare_protein_build_MD(int c_distancia) {
	uint tamanho_vetor = pdb_analise.size();
	int last_seq = -1;
	int cont_seq = 0;
	int cont_vector = -1;
	int i1c = 0;
	int i2c = c_distancia;
	GLfloat soma_total = 0.0;

	for (GLint i = 0; i < atomos_quantidade; i++) {
//		printf("3d\n");
		if (last_seq >= 0) {
			if (last_seq != amino_sequencial[i]) {
				cont_seq++;
				cont_vector = -1;
				if (cont_seq > i2c) {
					soma_total += compare_protein_calculateMD();
					cont_seq = 1; // antes tava 0
					i1c++;
					i2c++;
					i = 0;
				}
			}
		} else {
			last_seq = amino_sequencial[i];
		}
//		printf("Last seq: %d\n", last_seq);
		if (i1c == cont_seq) {
			cont_vector++;
			AminoTemp1MD.resize(cont_vector + 1);
			AminoTemp1MD[cont_vector].x = posx[i];
			AminoTemp1MD[cont_vector].y = posy[i];
			AminoTemp1MD[cont_vector].z = posz[i];
		}
		if (i2c == cont_seq) {
			if (i2c != i1c) {
				cont_vector++;
			}
			AminoTemp2MD.resize(cont_vector + 1);
			AminoTemp2MD[cont_vector].x = posx[i];
			AminoTemp2MD[cont_vector].y = posy[i];
			AminoTemp2MD[cont_vector].z = posz[i];
		}
		last_seq = amino_sequencial[i];
	}
	soma_total += compare_protein_calculate();
	printf("Total final1: %f\n", soma_total);
}

void compare_protein_build(int c_distancia) {
//	c_distancia++;
	uint tamanho_vetor = pdb_analise.size();
	printf("%d\n", tamanho_vetor);
	int last_seq = -1;
	int cont_seq = 0;
	int cont_vector = -1;
	int i1c = 0;
	int i2c = c_distancia;
	GLfloat soma_total = 0.0;
	for (uint i = 0; i < tamanho_vetor; ++i) {
		if (last_seq > 0) {
			if (last_seq != pdb_analise[i].aminoseq) {
				cont_seq++;
				cont_vector = -1;
				if (cont_seq > i2c) {
					soma_total += compare_protein_calculate();
					cont_seq = 1;
					i1c++;
					i2c++;
					i = 0;
				}
			}
		} else {
			last_seq = pdb_analise[i].aminoseq;
		}
//		printf("ContsetA %d\n",cont_seq);
		if (i1c == cont_seq) {
//			printf("Aqui 1: %d\n", cont_vector);
			cont_vector++;
			AminoTemp1.resize(cont_vector + 1);
			AminoTemp1[cont_vector].x = pdb_analise[i].x;
			AminoTemp1[cont_vector].y = pdb_analise[i].y;
			AminoTemp1[cont_vector].z = pdb_analise[i].z;
		}
//		printf("ContsetB %d\n",cont_seq);
		if (i2c == cont_seq) {
//			printf("Aqui 2\n");
			if (i2c != i1c) {
				cont_vector++;
			}
			AminoTemp2.resize(cont_vector + 1);
			AminoTemp2[cont_vector].x = pdb_analise[i].x;
			AminoTemp2[cont_vector].y = pdb_analise[i].y;
			AminoTemp2[cont_vector].z = pdb_analise[i].z;
		}
//		printf("ContsetC %d\n",cont_seq);
		last_seq = pdb_analise[i].aminoseq;
	}
	soma_total += compare_protein_calculate();
	printf("Total final2: %f\n", soma_total);
}

void load_protein_position(string PDBID) {
	printf("Lendo proteina para analise: %s\n", PDBID.c_str());
	try {
		sql::Driver *driver;
		sql::Connection *con;
		sql::PreparedStatement *pstmt;
		sql::ResultSet *res;
		driver = get_driver_instance();
		con = driver->connect("tcp://127.0.0.1:3306", "a00s_230", "testando");
		con->setSchema("a00s_230");
//		pstmt = con->prepareStatement("SELECT i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL ORDER BY i_306344,atomlabel DESC");
		pstmt = con->prepareStatement("SELECT i_306344 aminoseq, i_306299 x, i_306307 y, i_306315 z, i_331770 atomlabel FROM a_306280 WHERE i_307676=? AND i_306401=1 AND i_331770 IS NOT NULL AND i_306344=1 ORDER BY i_306344,atomlabel DESC");
		pstmt->setString(1, PDBID);
		res = pstmt->executeQuery();
		int cont_atom = -1;
		while (res->next()) {
			cont_atom++;
			pdb_analise.resize(cont_atom + 1);
			pdb_analise[cont_atom].aminoseq = res->getInt("aminoseq");
			pdb_analise[cont_atom].label = res->getString("atomlabel").c_str();
			pdb_analise[cont_atom].x = res->getDouble("x");
			pdb_analise[cont_atom].y = res->getDouble("y");
			pdb_analise[cont_atom].z = res->getDouble("z");
		}
		delete res;
		delete pstmt;
		delete con;
		printf("PDBid Loaded\n");
//		compare_protein_build_MD(0);
		compare_protein_build(0);
	} catch (sql::SQLException &e) {
		printf("%d\n", e.getErrorCode());
	}
}

