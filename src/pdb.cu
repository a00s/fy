#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
using namespace std;

vector<GLfloat> pdb_coordenadas_x;
vector<GLfloat> pdb_coordenadas_y;
vector<GLfloat> pdb_coordenadas_z;
GLint contador = 0;

void read_pdb() {
	printf("Lendo arquivo PDB\n");
	string line = "";
	ifstream myfile("files/Tunnel_4UG0.pdb");
	if (myfile.is_open()) {
		string tipo;
		GLfloat fix_x = 0;
		GLfloat fix_y = 0;
		GLfloat fix_z = 0;
		while (getline(myfile, line)) {
			tipo = line.substr(0, 6);
			if (tipo.erase(tipo.find_last_not_of(" ") + 1) == "ATOM" || tipo == "HETATM") {
				contador++;
				pdb_coordenadas_x.resize(contador);
				pdb_coordenadas_y.resize(contador);
				pdb_coordenadas_z.resize(contador);

				pdb_coordenadas_x[contador - 1] = strtof(line.substr(30, 8).c_str(), NULL) - fix_x;
				pdb_coordenadas_y[contador - 1] = strtof(line.substr(38, 8).c_str(), NULL) - fix_y;
				pdb_coordenadas_z[contador - 1] = strtof(line.substr(46, 8).c_str(), NULL) - fix_z;

				if (contador == 1) {
					fix_x = pdb_coordenadas_x[contador - 1];
					fix_y = pdb_coordenadas_y[contador - 1];
					fix_z = pdb_coordenadas_z[contador - 1];
					pdb_coordenadas_x[contador - 1] = 0.0;
					pdb_coordenadas_y[contador - 1] = 0.0;
					pdb_coordenadas_z[contador - 1] = 0.0;
				}

//				GLfloat cart_x = strtof(line.substr(30, 8).c_str(), NULL);
//				GLfloat cart_y = strtof(line.substr(38, 8).c_str(), NULL);
//				GLfloat cart_z = strtof(line.substr(46, 8).c_str(), NULL);
////				printf("%s\n",line.c_str());
//				printf("%f %f %f\n", cart_x, cart_y, cart_z);

//				glColor3ub(255, 255, 255);
//				glTranslatef(0.0, 0.0, 0.0);
//				glutSolidSphere(0.1f, 6, 6);
//				glTranslatef(0.0, 0.0, 0.0);
			}
		}
		myfile.close();
	} else {
		cout << "Unable to open file\n";
	}
}

void add_pdb() {
	for (int i = 0; i < contador; i++) {
		glColor3ub(255, 255, 255);
		glTranslatef(pdb_coordenadas_x[i], pdb_coordenadas_y[i], pdb_coordenadas_z[i]);
		glutSolidCube(0.1f);
		glTranslatef(-pdb_coordenadas_x[i], -pdb_coordenadas_y[i], -pdb_coordenadas_z[i]);
	}
}

void grava_pdb() {
	printf("Lendo arquivo PDB\n");
	//Grava
	ofstream myfile("example.txt");
	if (myfile.is_open()) {
		myfile << "This is a line.\n";
		myfile << "This is another line.\n";
		myfile.close();
	} else
		cout << "Unable to open file";
}
