#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>
using namespace std;

GLint max_contador = 0;
extern GLfloat max_distance_hydrogen_bond;
extern GLint forca_externa_contador_max_hb;

extern GLint forca_externa_contador_max;
GLint forca_externa_contador_max_best = 0;

extern GLint forca_externa_contador_max_t;
GLint forca_externa_contador_max_best_t = 0;

extern GLint calibration_minimal_distance;

extern GLint contador_restart_life;

void change_properties(){

	// Procurando valores pra tracao e colisao ------------------------------------------------
	if(contador_restart_life > max_contador){
		printf("------------- Encontrado menor ------------- %d %d\n",contador_restart_life,max_contador);
		forca_externa_contador_max_best = forca_externa_contador_max;
		forca_externa_contador_max_best_t = forca_externa_contador_max_t;
		max_contador = contador_restart_life;
	}
	forca_externa_contador_max++;
	if(forca_externa_contador_max > 30){
		forca_externa_contador_max = 0;
		forca_externa_contador_max_t++;
		if(forca_externa_contador_max_t > 30){
				forca_externa_contador_max_t = 0;
		}
	}
	printf("forca_externa_contador_max: %d t: %d best: %d bestt: %d max_contador_best: %d\n",forca_externa_contador_max,forca_externa_contador_max_t,forca_externa_contador_max_best,forca_externa_contador_max_best_t,max_contador);
	// ----------------------------------------------------------------------------------------
}
