#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/GLee.h>         // No need to link to GL/gl.h
#include <GL/glfw.h>      // Include OpenGL Framework library
#include <GL/freeglut.h>  // Include FreeGLUT so we can easily draw spheres and calculate our viewing frustrum
#include <math.h>         // Used only for sin() and cos() functions
#include <cstdio>
#include <stdlib.h>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <string>
#include <vector>
#include <map>
using namespace std;

GLint max_contador = 0;
extern map<string, GLfloat> vs;
extern map<string, map<string, map<string, map<string, map<string, map<string, GLfloat> > > > > > vp;

//extern GLfloat max_distance_hydrogen_bond;
extern GLint forca_externa_contador_max_hb;

extern GLint forca_externa_contador_max;
GLint forca_externa_contador_max_best = 0;

extern GLint forca_externa_contador_max_t;
GLint forca_externa_contador_max_best_t = 0;

extern GLint calibration_minimal_distance;

extern GLint contador_restart_life;

GLint max_life = 0;
GLfloat best_valueF = 0;
GLint best_valueI = 0;

extern void sV(string s1, string s2, GLfloat valor);
extern GLfloat gV(string s1, string s2);

extern map<string, map<string, GLfloat*> > vF;
extern map<string, map<string, GLint*> > vI;

void change_properties() {
//	printf("V1==> %f\n",*vF["tuning"]["max_distance_hydrogen_bond"]);
//	if(contador_restart_life > max_life){
//		best_valueF = *vF["tuning"]["max_distance_hydrogen_bond"];
//		max_life = contador_restart_life;
//	}
//	*vF["tuning"]["max_distance_hydrogen_bond"] += 0.1;
//	printf("V2==> %f (%f / %d)\n",*vF["tuning"]["max_distance_hydrogen_bond"], best_valueF, max_life);


//	printf("V1==> %d\n", *vI["tuning"]["forca_externa_contador_max"]);
	if (contador_restart_life > max_life) {
		best_valueI = *vI["tuning"]["forca_externa_contador_max"];
		max_life = contador_restart_life;
	}
	*vI["tuning"]["forca_externa_contador_max"] += 1;
	printf("V2==> %d (%d / %d)\n", *vI["tuning"]["forca_externa_contador_max"], best_valueI, max_life);

//	for (map<string, map<string, GLfloat*> >::iterator i = vF.begin(); i != vF.end(); ++i) {
//			for (map<string, GLfloat*>::iterator ii = i->second.begin(); ii != i->second.end(); ++ii) {
//				printf("Variavel %s %f\n",(*ii).first,(*ii).second);
////				string campochave = (*ii).first;
////				tw[campochave] = Convert((*ii).second);
//			}
//	}

// Procurando valores pra tracao e colisao ------------------------------------------------
//	if(contador_restart_life > max_contador){
//		printf("------------- Encontrado menor ------------- %d %d\n",contador_restart_life,max_contador);
//		forca_externa_contador_max_best = forca_externa_contador_max;
//		forca_externa_contador_max_best_t = forca_externa_contador_max_t;
//		max_contador = contador_restart_life;
//	}
//	forca_externa_contador_max++;
//	if(forca_externa_contador_max > 30){
//		forca_externa_contador_max = 0;
//		forca_externa_contador_max_t++;
//		if(forca_externa_contador_max_t > 30){
//			forca_externa_contador_max_t = 0;
//		}
//	}
//	printf("forca_externa_contador_max: %d %f: %d best: %d bestt: %d max_contador_best: %d\n",forca_externa_contador_max,forca_externa_contador_max_t,forca_externa_contador_max_best,forca_externa_contador_max_best_t,max_contador);
	// ----------------------------------------------------------------------------------------
}
